
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

/* device data taken from CUDA occupancy calculator */

#ifdef __CUDA_ARCH__

/* 2.0 and 2.1 */
#if __CUDA_ARCH__ == 200 || __CUDA_ARCH__ == 210
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 32768
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 8
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 32
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 40

/* 3.0 and 3.5 */
#elif __CUDA_ARCH__ == 300 || __CUDA_ARCH__ == 350
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 63
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 3.2 */
#elif __CUDA_ARCH__ == 320
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 32768
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 63
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 3.7 */
#elif __CUDA_ARCH__ == 370
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 255

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 63
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 5.0, 5.2, 5.3, 6.0, 6.1 */
#elif __CUDA_ARCH__ >= 500
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 32
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 255

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 48
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* unknown architecture */
#else
#  error "Unknown or unsupported CUDA architecture, can't determine launch bounds"
#endif

#include "../../kernel_compat_cuda.h"
#include "../../kernel_math.h"
#include "../../kernel_types.h"
#include "../../kernel_globals.h"
#include "../../kernel_film.h"
#include "../../kernel_path.h"
#include "../../kernel_path_branched.h"
#include "../../kernel_bake.h"

#include "../../filter/filter.h"

/* compute number of threads per block and minimum blocks per multiprocessor
 * given the maximum number of registers per thread */

#define CUDA_LAUNCH_BOUNDS(threads_block_width, thread_num_registers) \
	__launch_bounds__( \
		threads_block_width*threads_block_width, \
		CUDA_MULTIPRESSOR_MAX_REGISTERS/(threads_block_width*threads_block_width*thread_num_registers) \
		)

/* sanity checks */

#if CUDA_THREADS_BLOCK_WIDTH*CUDA_THREADS_BLOCK_WIDTH > CUDA_BLOCK_MAX_THREADS
#  error "Maximum number of threads per block exceeded"
#endif

#if CUDA_MULTIPRESSOR_MAX_REGISTERS/(CUDA_THREADS_BLOCK_WIDTH*CUDA_THREADS_BLOCK_WIDTH*CUDA_KERNEL_MAX_REGISTERS) > CUDA_MULTIPROCESSOR_MAX_BLOCKS
#  error "Maximum number of blocks per multiprocessor exceeded"
#endif

#if CUDA_KERNEL_MAX_REGISTERS > CUDA_THREAD_MAX_REGISTERS
#  error "Maximum number of registers per thread exceeded"
#endif

#if CUDA_KERNEL_BRANCHED_MAX_REGISTERS > CUDA_THREAD_MAX_REGISTERS
#  error "Maximum number of registers per thread exceeded"
#endif

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_branched_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_shader(uint4 *input,
                   float4 *output,
                   float *output_luma,
                   int type,
                   int sx,
                   int sw,
                   int offset,
                   int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		kernel_shader_evaluate(NULL,
		                       input,
		                       output,
		                       output_luma,
		                       (ShaderEvalType)type, 
		                       x,
		                       sample);
	}
}

#ifdef __BAKING__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(uint4 *input, float4 *output, int type, int filter, int sx, int sw, int offset, int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw)
		kernel_bake_evaluate(NULL, input, output, (ShaderEvalType)type, filter, x, offset, sample);
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_shadow(int sample, float* buffers, int4 buffer_rect, int offset, int stride, float *unfiltered, float *sampleVariance, float *sampleVarianceV, float *bufferVariance, int4 prefilter_rect)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		int tile_x[4] = {buffer_rect.x, buffer_rect.x, buffer_rect.x+buffer_rect.z, buffer_rect.x+buffer_rect.z};
		int tile_y[4] = {buffer_rect.y, buffer_rect.y, buffer_rect.y+buffer_rect.w, buffer_rect.y+buffer_rect.w};
		float *tile_buffers[9] = {NULL, NULL, NULL, NULL, buffers, NULL, NULL, NULL, NULL};
		int tile_offset[9] = {0, 0, 0, 0, offset, 0, 0, 0, 0};
		int tile_stride[9] = {0, 0, 0, 0, stride, 0, 0, 0, 0};
		kernel_filter_divide_shadow(NULL, sample, tile_buffers, x, y, tile_x, tile_y, tile_offset, tile_stride, unfiltered, sampleVariance, sampleVarianceV, bufferVariance, prefilter_rect);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_get_feature(int sample, float* buffers, int m_offset, int v_offset, int4 buffer_rect, int offset, int stride, float *mean, float *variance, int4 prefilter_rect)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		int tile_x[4] = {buffer_rect.x, buffer_rect.x, buffer_rect.x+buffer_rect.z, buffer_rect.x+buffer_rect.z};
		int tile_y[4] = {buffer_rect.y, buffer_rect.y, buffer_rect.y+buffer_rect.w, buffer_rect.y+buffer_rect.w};
		float *tile_buffers[9] = {NULL, NULL, NULL, NULL, buffers, NULL, NULL, NULL, NULL};
		int tile_offset[9] = {0, 0, 0, 0, offset, 0, 0, 0, 0};
		int tile_stride[9] = {0, 0, 0, 0, stride, 0, 0, 0, 0};
		kernel_filter_get_feature(NULL, sample, tile_buffers, m_offset, v_offset, x, y, tile_x, tile_y, tile_offset, tile_stride, mean, variance, prefilter_rect);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_combine_halves(float *mean, float *variance, float *a, float *b, int4 prefilter_rect, int r)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_combine_halves(x, y, mean, variance, a, b, prefilter_rect, r);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_construct_transform(int sample, float const* __restrict__ buffer, float *transform, void *storage, int4 filter_area, int4 rect)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		float *l_transform = transform + y*filter_area.z + x;
		kernel_filter_construct_transform(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, l_transform, l_storage, rect, filter_area.z*filter_area.w, threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_combined(float *buffers, int sample, int offset, int stride, int4 filter_area)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		kernel_filter_divide_combined(NULL, x + filter_area.x, y + filter_area.y, sample, buffers, offset, stride);
	}
}


extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_difference(int dx, int dy, float ccl_readonly_ptr weightImage, float ccl_readonly_ptr varianceImage, float *differenceImage, int4 rect, int w, float a, float k_2) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_calc_difference(x, y, dx, dy, weightImage, varianceImage, differenceImage, rect, w, 0, a, k_2);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_blur(float ccl_readonly_ptr differenceImage, float *outImage, int4 rect, int w, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_blur(x, y, differenceImage, outImage, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_weight(float ccl_readonly_ptr differenceImage, float *outImage, int4 rect, int w, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_calc_weight(x, y, differenceImage, outImage, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_update_output(int dx, int dy, float ccl_readonly_ptr differenceImage, float ccl_readonly_ptr image, float *outImage, float *accumImage, int4 rect, int w, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_update_output(x, y, dx, dy, differenceImage, image, outImage, accumImage, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_normalize(float *outImage, float ccl_readonly_ptr accumImage, int4 rect, int w) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_normalize(x, y, outImage, accumImage, rect, w);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_construct_gramian(int dx, int dy, float ccl_readonly_ptr differenceImage, float ccl_readonly_ptr buffer, int color_pass, int variance_pass, void *storage, float const* __restrict__ transform, float *XtWX, float3 *XtWY, int4 rect, int4 filter_rect, int w, int h, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + max(0, rect.x-filter_rect.x);
	int y = blockDim.y*blockIdx.y + threadIdx.y + max(0, rect.y-filter_rect.y);
	if(x < min(filter_rect.z, rect.z-filter_rect.x) && y < min(filter_rect.w, rect.w-filter_rect.y)) {
		kernel_filter_nlm_construct_gramian(x, y, dx, dy, differenceImage, buffer, color_pass, variance_pass, (CUDAFilterStorage*) storage, transform, XtWX, XtWY, rect, filter_rect, w, h, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_finalize(int w, int h, float *buffer, void *storage, float *XtWX, float3 *XtWY, int4 filter_area, int4 buffer_params, int sample) {
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		kernel_filter_finalize(x+filter_area.x, y+filter_area.y, y*filter_area.z+x, filter_area.z*filter_area.w, w, h, buffer, (CUDAFilterStorage*) storage, XtWX, XtWY, buffer_params, sample);
	}
}

#endif

